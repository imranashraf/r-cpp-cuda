#include <cmath>
#include <stdio.h>
#include <hip/hip_runtime.h>
// #include "cuda_runtime.h"

__global__ void
vectorInc(double *A, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i < numElements)
    {
        A[i] = A[i] + 1;
    }
}

extern "C"
void cudaCall(double *A, int n)
{
    printf("Halo van CUDA\n");

    // Device Memory
    double *d_A;
    // Define the execution configuration
    dim3 blockSize(256,1,1);
    dim3 gridSize(1,1,1);
    gridSize.x = (n + blockSize.x - 1) / blockSize.x;

    // Allocate output array
    hipMalloc((void**)&d_A, n * sizeof(double));

    // Copy data to device
    hipMemcpy(d_A, A, n * sizeof(double), hipMemcpyHostToDevice);

    // Call the kernel
    vectorInc<<<gridSize,blockSize>>>(d_A, n);

    // Copy output from device to host
    hipMemcpy(A, d_A, n * sizeof(double), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_A);
}
